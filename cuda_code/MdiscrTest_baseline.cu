#include "MdiscrHost_baseline.cu.h"
#include "HelpersHost.cu.h"
#include "MainCommon.h"

#include <stdio.h>
//#include "hip/hip_runtime_api.h"

template<class ModN>
int test(const unsigned int num_elems) {
    
    // Allocate memory.
    typename ModN::InType* h_in =
        (typename ModN::InType*) malloc(num_elems * sizeof(typename ModN::InType));
    typename ModN::InType* h_out =
        (typename ModN::InType*) malloc(num_elems * sizeof(typename ModN::InType));
    unsigned int* h_out_sizes = (unsigned int*) malloc(num_elems * sizeof(unsigned int));
    
    { // Initialize array.
        
        // Seed the random number generator.
        std::srand(time(NULL));
        
        populateIntArray(num_elems, h_in);
    }
    /* printIntArray(num_elems, "h_in", h_in); */
    
    typename ModN::InType *d_in, *d_out;
    unsigned int *d_out_sizes;
    { // Device allocation.
        hipMalloc((void**)&d_in ,   num_elems * sizeof(typename ModN::InType));
        hipMalloc((void**)&d_out,   num_elems * sizeof(typename ModN::InType));
        hipMalloc((void**)&d_out_sizes, num_elems * sizeof(unsigned int));
        
        // Copy host memory to device.
        hipMemcpy(d_in, h_in, num_elems * sizeof(typename ModN::InType), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    }
    
    /* hipProfilerStart(); */
    
    // Call the discriminator function.
    mdiscr<ModN>(num_elems, d_in, d_out, d_out_sizes);
    //typename ModN::TupleType sizes = mdiscr<ModN>(num_elems, (1<<16), d_in, d_out);
    
    /* hipProfilerStop(); */
    
    // Copy result back to host.
    hipMemcpy(h_out, d_out, num_elems * sizeof(typename ModN::InType), hipMemcpyDeviceToHost);
    hipMemcpy(h_out_sizes, d_out_sizes, num_elems * sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    // Free device memory.
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_sizes);
    
    /* printIntArray(num_elems, "h_out", h_out); */
    /* printIntArray(num_elems, "h_out_sizes", h_out_sizes); */
    
    bool success = validateOneSegment<ModN>(h_in, h_out, h_out_sizes, num_elems);
    
    // Cleanup memory.
    free(h_in);
    free(h_out);
    free(h_out_sizes);
    
    if (success) {
        fprintf(stderr, "mdiscr on %d elems: VALID RESULT!\n", num_elems);
        return EXIT_SUCCESS;
    }
    else {
        fprintf(stderr, "mdiscr on %d elems: INVALID RESULT!\n", num_elems);
        return EXIT_FAILURE;
    }
}
