#include "MdiscrHost_optimized.cu.h"
#include "HelpersHost.cu.h"
#include "MainCommon.h"

#include <stdio.h>
//#include "hip/hip_runtime_api.h"

/*
 * This version of the program uses the non-compact representation of the
 * tuple as default. To use the compact representation, the flag
 * COMPACT_REPRESENTATION must be set. Flags/constants can be set/defined,
 * e.g., on the command-line using the -D flag. To run the program, these
 * constants also need to be defined:
 *   - MAX_CHUNK
 *   - MAP_X
 *   - MAP_Y
 *   - WRITE_X
 *   - WRITE_Y
 *
 * The default is to use Mod4 as the discriminator. If NUM_CLASSES=X is defined,
 * then Mod<X> is used, but in that case we need to define PACKED_VY where Y is
 * either 1, 2, or 3.
 *
 * Also, these options can be given at the command-line:
 *   - optimization level, e.g., -O3 (optional)
 *   - NDEBUG (optional)
 *   - arch=sm_20 (optional, eliminate warnings)
 */

template<class ModN>
int test(const unsigned int num_elems) {
    fprintf(stderr, "sizeof(typename ModN::TupleType) = %d\n", sizeof(typename ModN::TupleType));
    fprintf(stderr, "sizeof(typename ModN::TupleType::SmallType) = %d\n", sizeof(typename ModN::TupleType::SmallType));
    fprintf(stderr, "sizeof(typename ModN::TupleType::MediumType) = %d\n", sizeof(typename ModN::TupleType::MediumType));
    
    // Allocate memory.
    typename ModN::InType* h_in =
        (typename ModN::InType*) malloc(num_elems * sizeof(typename ModN::InType));
    typename ModN::InType* h_out =
        (typename ModN::InType*) malloc(num_elems * sizeof(typename ModN::InType));
    unsigned int* h_out_sizes = (unsigned int*) malloc(num_elems * sizeof(unsigned int));
    
    { // Initialize array.
        
        // Seed the random number generator.
        std::srand(time(NULL));
        
        populateIntArray(num_elems, h_in);
    }
#ifdef PRINT
    printIntArray(num_elems, "h_in", h_in);
#endif
    
    typename ModN::InType *d_in, *d_out;
    unsigned int *d_out_sizes;
    { // Device allocation.
        hipMalloc((void**)&d_in ,   num_elems * sizeof(typename ModN::InType));
        hipMalloc((void**)&d_out,   num_elems * sizeof(typename ModN::InType));
        hipMalloc((void**)&d_out_sizes, num_elems * sizeof(unsigned int));
        
        // Copy host memory to device.
        hipMemcpy(d_in, h_in, num_elems * sizeof(typename ModN::InType), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    }
    
    /* hipProfilerStart(); */
    
    // Call the discriminator function.
    typename ModN::TupleType sizes = mdiscr<ModN>(num_elems, (1<<16), d_in, d_out);
    
    /* hipProfilerStop(); */
    
    // Copy result back to host.
    hipMemcpy(h_out, d_out, num_elems * sizeof(typename ModN::InType), hipMemcpyDeviceToHost);
    hipMemcpy(h_out_sizes, d_out_sizes, num_elems * sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    // Free device memory.
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_sizes);
    
#ifdef PRINT
    printIntArray(num_elems, "h_out", h_out);
    /* printIntArray(num_elems, "h_out_sizes", h_out_sizes); */
#endif
    
    bool success = validateOneSegment<ModN>(h_in, h_out, sizes, num_elems);
    
    if (success) {
        fprintf(stderr, "mdiscr on %d elems: VALID RESULT!\n", num_elems);
    }
    else {
        fprintf(stderr, "mdiscr on %d elems: INVALID RESULT!\n", num_elems);
    }
    
    // Cleanup memory.
    free(h_in);
    free(h_out);
    free(h_out_sizes);
    
    if (success) {
        return EXIT_SUCCESS;
    }
    else {
        return EXIT_FAILURE;
        
        // This generates a core dump (run "ulimit -c unlimited" beforehand to
        // allow core dumps).
        // abort();
    }
}
