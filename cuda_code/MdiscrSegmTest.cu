#include "MdiscrSegmHost.cu.h"
#include "HelpersHost.cu.h"
#include "MainCommon.h"

#include <stdio.h>
#include <cassert>

template<class ModN>
bool validate(typename ModN::InType*  h_in,
              int*                    h_in_sizes,
              typename ModN::InType*  h_out,
              int*                    h_out_sizes,
              unsigned int            num_elems
    ) {
    
    bool success = true;
    unsigned int i = 0;
    unsigned int j = 0;
    int size;
    
    while (i < num_elems) {
        // Size of the current segment.
        size = h_in_sizes[i];
        
        // We are only accessing segment starts, all sizes should be non-zero.
        assert(size != 0);
        
        // Validate one segment at a time.
        //printf("Validating segment number %d with size %d...\n", j, size);
        if (!validateOneSegment<ModN>(&h_in[i], &h_out[i], &h_out_sizes[i], size)) {
            success = false;
        }
        
        // Jump to the next segment start.
        i+=size;
        
        j++;
    }
    
    return success;
}

template<class ModN>
void test(const unsigned int num_elems
    ) {
    
    // Allocate memory.
    typename ModN::InType* h_in  =
        (typename ModN::InType*) malloc(num_elems * sizeof(typename ModN::InType));
    typename ModN::InType* h_out =
        (typename ModN::InType*) malloc(num_elems * sizeof(typename ModN::InType));
    int* h_in_sizes  = (int*) malloc(num_elems * sizeof(int));
    int* h_out_sizes = (int*) malloc(num_elems * sizeof(int));
    
    { // Initialize array.
        
        // Seed the random number generator.
        std::srand(time(NULL));
        
        populateIntArray(num_elems, h_in);
        populateSizesArray(num_elems, h_in_sizes);
    }
    //printIntArray(num_elems, "h_in", h_in);
    //printIntArray(num_elems, "h_in_sizes", h_in_sizes);
    
    typename ModN::InType *d_in, *d_out;
    int *d_in_sizes, *d_out_sizes;
    { // Device allocation.
        hipMalloc((void**)&d_in ,       num_elems * sizeof(typename ModN::InType));
        hipMalloc((void**)&d_out,       num_elems * sizeof(typename ModN::InType));
        hipMalloc((void**)&d_in_sizes,  num_elems * sizeof(int));
        hipMalloc((void**)&d_out_sizes, num_elems * sizeof(int));
        
        // Copy host memory to device.
        hipMemcpy(d_in, h_in, num_elems * sizeof(typename ModN::InType), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        hipMemcpy(d_in_sizes, h_in_sizes, num_elems * sizeof(int), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    }
    
    // Call the discriminator function.
    mdiscrSegm<ModN>(num_elems, d_in, d_in_sizes, d_out, d_out_sizes);
    
    // Copy result back to host.
    hipMemcpy(h_out, d_out, num_elems * sizeof(typename ModN::InType), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipMemcpy(h_out_sizes, d_out_sizes, num_elems * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    // Free device memory.
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_in_sizes);
    hipFree(d_out_sizes);
    
    //printIntArray(num_elems, "h_out", h_out);
    //printIntArray(num_elems, "h_out_sizes", h_out_sizes);
    
    bool success = validate<ModN>(h_in, h_in_sizes, h_out, h_out_sizes, num_elems);
    
    if (success) {
        printf("mdiscrSegm on %d elems: VALID RESULT!\n", num_elems);
    }
    else {
        printf("mdiscrSegm on %d elems: INVALID RESULT!\n", num_elems);
    }
    
    // Cleanup memory.
    free(h_in);
    free(h_out);
    free(h_in_sizes);
    free(h_out_sizes);
}
